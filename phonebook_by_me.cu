
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

__global__ void searchContact(char* contact_string, char* name, int* index, int contact_len, int name_len, int no_of_contact, int no_of_core)
{
    int idx = threadIdx.x;
    int start_index = idx*(no_of_contact/no_of_core);
    int end_index = (idx+1)*(no_of_contact/no_of_core);
    if(idx == no_of_core-1) end_index = contact_len;
    else end_index = index[end_index];

    int hash_count = start_index;
    int find = 0;
    for(int i=index[start_index]; i<end_index; i++)
    {
        if(contact_string[i]=='#') 
        {
            find = 0;
            hash_count++;
        }
        if(contact_string[i]==name[find]) find++;
        else find = 0;
        if(find == name_len) 
        {
            find = 0;
            index[hash_count] = -1;
        }
    }
}

__global__ void searchContact2(char *C, char *N, int *index, int contact_len, int name_len, int number_of_core, int number_of_contacts)
{
    int idx = threadIdx.x;
    int start_index = idx*(number_of_contacts/number_of_core);
    int end_index = (idx+1)*(number_of_contacts/number_of_core);
    if(idx==number_of_core-1) end_index = contact_len;
    else end_index = index[end_index];
    int find = 0;
    int hash_count = start_index;
    for(int i=index[start_index]; i<end_index; i++)
    {
        if((C[i]=='#'))
        {
            find = 0;
            hash_count++;
        }
        if(C[i]==N[find]) find++;
        else find = 0;
        if(find==name_len)
        {
            index[hash_count] = -1;
            find = 0;
        }
    }
}


int main(int argc, char* argv[])
{
    string name;
    cout << "Enter the name to search : ";
    cin >> name;
    for(auto &c : name) c = toupper(c);

    vector<string>contact_list, original_list;
    vector<string>file_names(argv+1, argv+argc);
    for(auto file_name : file_names)
    {
        ifstream file(file_name);
        string line;
        while(getline(file, line))
        {
            original_list.push_back(line);
            for(auto &c : line) c = toupper(c);
            contact_list.push_back(line);
        }
        file.close();
    }

    string contact_string = "";
    int number_of_contact = original_list.size();
    vector<int>contact_index(number_of_contact);
    for(int i=0; i<number_of_contact; i++)
    {
        contact_index[i] = contact_string.size();
        contact_string+=contact_list[i]+"#";
    }

    cout << "Enter the nubmer of core : ";
    int no_of_core;
    cin >> no_of_core;

    /// alocating memory
    char *DC, *DN;
    int *index;
    int contact_len = contact_string.size()+1;
    int name_len = name.size();
    hipMalloc(&DC, contact_len);
    hipMemcpy(DC, &contact_string[0], contact_len, hipMemcpyHostToDevice);
    hipMalloc(&DN, name_len);
    hipMemcpy(DN, &name[0], name_len, hipMemcpyHostToDevice);
    hipMalloc(&index, number_of_contact*sizeof(int));
    hipMemcpy(index, &contact_index[0], number_of_contact*sizeof(int), hipMemcpyHostToDevice);


    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    //searchContact<<<1, no_of_core>>>(DC, DN, index, contact_len, name_len, number_of_contact, no_of_core);
    searchContact2<<<1, no_of_core>>>(DC, DN, index, contact_len, name_len, no_of_core, number_of_contact);

    hipDeviceSynchronize();
    hipEventRecord(end);
    hipEventSynchronize(end);
    float miliseconds = 0;
    hipEventElapsedTime(&miliseconds, start, end);
    cout << "Time Taken : " << miliseconds << endl;

    vector<int>res(number_of_contact);
    hipMemcpy(&res[0], index, number_of_contact*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<number_of_contact; i++)
    {
        if(res[i]==-1) cout << original_list[i] << endl;
    }
    cout << res.size() << endl;

}