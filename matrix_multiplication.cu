
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;

__global__ void matrix_multiplication(int* A, int* B, int* C, int* DP)
{
    int N = DP[0];
    int M = DP[1];
    int P = DP[2];
    int number_of_matrix = DP[3];
    int number_of_core = DP[4];

    int i = threadIdx.x;
    int start_matrix_index = (number_of_matrix/number_of_core)*i;
    int end_matrix_index = (number_of_matrix/number_of_core)*(i+1);
    if(i==number_of_core-1) end_matrix_index = number_of_matrix;

    for(int x = start_matrix_index; x<end_matrix_index; x++)
    {
        int addA = x*N*M;
        int addB = x*M*P;
        int addC = x*N*P;

        for(int i=0; i<N; i++)
        {
            for(int j=0; j<P; j++)
            {
                int sum = 0;
                for(int k=0; k<M; k++)
                {
                    sum += A[addA + i*M + k] * B[addB + k*P + j];
                }
                C[addC + i*P+j] = sum;
            }
        }
    }
}

int main(int argc, char *argv[])
{
    cout << "Enter the dimension of the matrix N, M, P : " << endl;
    int N, M, P;
    cin >> N >> M >> P;
    cout << "Enter the number of matrix : " << endl;
    int number_of_matrix;
    cin >> number_of_matrix;
    
    int *A, *B, *C;
    int a_element = number_of_matrix*N*M;
    int b_element = number_of_matrix*M*P;
    int c_element = number_of_matrix*N*P;

    A = new int[a_element];
    B = new int[b_element];
    C = new int[c_element];

    for(int i=0; i<a_element; i++)
    {
        A[i] = 1;
    }
    for(int i=0; i<b_element; i++)
    {
        B[i] = 1;
    }

    int *DA, *DB, *DC;
    hipMalloc(&DA, a_element*sizeof(int));
    hipMalloc(&DB, b_element*sizeof(int));
    hipMalloc(&DC, c_element*sizeof(int));

    hipMemcpy(DA, A, a_element*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(DB, B, b_element*sizeof(int), hipMemcpyHostToDevice);


    cout << "Enter the number of core you want to use : " << endl;
    int number_of_core;
    cin >> number_of_core;

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    int parameter[] = {N, M, P, number_of_matrix, number_of_core};
    int *DP;
    hipMalloc(&DP, 5*sizeof(int));
    hipMemcpy(DP, parameter, 5*sizeof(int), hipMemcpyHostToDevice);

    matrix_multiplication <<< 1, number_of_core >>> (DA, DB, DC, DP);

    hipDeviceSynchronize();

    hipEventRecord(end);
    hipEventSynchronize(end);

    float milisecond = 0;
    hipEventElapsedTime(&milisecond, start, end);
    cout << "Time taken with " << number_of_core << " is : " << milisecond << " ms" << endl;

    hipMemcpy(C, DC, c_element*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++)
    {
        for(int j=0; j<P; j++)
        {
            cout << C[i*P+j] << " ";
        }
        cout << endl;
    }

    return 0;
}